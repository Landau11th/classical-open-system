#include "hip/hip_runtime.h"
/*
* This program uses the host CURAND API to generate 100
* pseudorandom floats .
*/

//C or C++ headers
#include <stdio.h>
#include <stdlib.h>
#include <ctime>
#include<iostream>

//CUDA headers
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include ""

//my headers
#include "rand64.hpp"

#define CUDA_CALL (x) do{ if (( x) != hipSuccess ) { printf (" Error at % s :% d\ n" , __FILE__ , __LINE__ ) ;return EXIT_FAILURE ;}}while(0)

#define CURAND_CALL (x) do{ if (( x) != HIPRAND_STATUS_SUCCESS ) { printf (" Error at % s :% d\ n" , __FILE__ , __LINE__ ) ;return EXIT_FAILURE ;}}while(0)


__global__ void natural_numbers(float* x_device, int length)
{
	int thrd_id = threadIdx.x + blockIdx.x * blockDim.x;

	while (thrd_id < length)
	{
		x_device[thrd_id] = thrd_id;
		thrd_id += blockDim.x* gridDim.x;
	}
}

__global__ void zeros(float* x_device, int length)
{
	int thrd_id = threadIdx.x + blockIdx.x * blockDim.x;

	while (thrd_id < length)
	{
		x_device[thrd_id] = 0.0f;
		thrd_id += blockDim.x* gridDim.x;
	}
}

__global__ void ones(float* x_device, int length)
{
	int thrd_id = threadIdx.x + blockIdx.x * blockDim.x;

	while (thrd_id < length)
	{
		x_device[thrd_id] = 1.0f;
		thrd_id += blockDim.x* gridDim.x;
	}
}

float f_integral(const float t)
{
	return sin(6.28318530718*t);
}

int main(int argc, char * argv[])
{
	size_t n = 1024*8*8;
	size_t N_t = 1024 * 16;
	float tau = 1.0;
	float dt = tau / N_t;
	float dt_sq = sqrt(dt);

	size_t i;
	hiprandGenerator_t gen;
	float *x_host, *y_host;
	float *x_device, *y_device, *rand_device;

	hipEvent_t start_cuda, stop_cuda;
	hipEventCreate(&start_cuda);
	hipEventCreate(&stop_cuda);
	float ellapese_time;

	
	//Allocate n floats on host
	x_host = (float *)calloc(n, sizeof(float));
		
	// Allocate n floats on device
	hipMalloc((void **)& x_device, n * sizeof(float));
	hipMalloc((void **)& rand_device, n * sizeof(float));
	


	//Assign x on device
	//natural_numbers<<< 2, 128 >>>(x_device, n);
	zeros<<<2, 256>>>(x_device, n);
	
	//Create pseudo - random number generator
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	//Set seed
	time_t seed;
	time(&seed);
	hiprandSetPseudoRandomGeneratorSeed(gen, seed);
	printf("Start random walk\n");
	hipEventRecord(start_cuda, 0);
	for (size_t i = 0; i < N_t; ++i)
	{
		//Generate n floats on device 
		//hiprandGenerateUniform(gen, rand_device, n);
		hiprandGenerateNormal(gen, rand_device, n, 0, f_integral((i*tau) / N_t)*dt_sq);
		//vector addition
		hipblasSaxpy(n, 1.0, rand_device, 1, x_device, 1);
	}
	hipEventRecord(stop_cuda, 0);
	hipEventSynchronize(stop_cuda);
	hipEventElapsedTime(&ellapese_time, start_cuda, stop_cuda);
	std::cout << ellapese_time << " ms\n";

	ones<<<2, 128>>>(rand_device, n);

	float average;
	std::cout << hipblasSdot(n, rand_device, 1, x_device, 1)/n << std::endl;
	std::cout << hipblasSdot(n, x_device, 1, x_device, 1)/n << std::endl;
	
	Deng::RandNumGen::LCG64 rand64(0);
	std::clock_t start;
	double duration;

	float temp;
	start = std::clock();
	for (size_t i = 0; i < N_t; ++i)
	{
		for (size_t j = 0; j < n; ++j)
		{
			temp = rand64();
			x_host[j] += temp;
		}
	}
	duration = (std::clock() - start) / (double)CLOCKS_PER_SEC;
	std::cout << "CPU costs " << 1000* duration << " ms\n";
	
	/* Copy device memory to host */
	hipMemcpy(x_host, x_device, n * sizeof(float), hipMemcpyDeviceToHost);
	//// Show result
	//for (i = 0; i < n; i++) {
	//	if (i % 8 != 0)
	//		printf(", ");
	//	else
	//		printf("\n");

	//	printf("%1.7f", x_host[i]);
	//}
	//printf("\n");
	// Cleanup
	hiprandDestroyGenerator(gen);
	hipFree(x_device);
	hipFree(rand_device);
	free(x_host);

	return EXIT_SUCCESS;
}