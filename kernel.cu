/*
* This program uses the host CURAND API to generate 100
* pseudorandom floats .
*/

//C or C++ headers
#include <stdio.h>
#include <stdlib.h>
#include <ctime>


//CUDA headers
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>

#define CUDA_CALL (x) do{ if (( x) != hipSuccess ) { printf (" Error at % s :% d\ n" , __FILE__ , __LINE__ ) ;return EXIT_FAILURE ;}}while(0)

#define CURAND_CALL (x) do{ if (( x) != HIPRAND_STATUS_SUCCESS ) { printf (" Error at % s :% d\ n" , __FILE__ , __LINE__ ) ;return EXIT_FAILURE ;}}while(0)




int main(int argc, char * argv[])
{
	size_t n = 100;
	size_t i;
	hiprandGenerator_t gen;
	float * devData, *hostData;
	/* Allocate n floats on host */
	hostData = (float *)calloc(n, sizeof(float));
	/* Allocate n floats on device */
	hipMalloc((void **)& devData, n * sizeof(float));
	/* Create pseudo - random number generator */
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	/* Set seed */
	time_t seed;
	time(&seed);
	hiprandSetPseudoRandomGeneratorSeed(gen, seed);
	/* Generate n floats on device */
	hiprandGenerateUniform(gen, devData, n);
	/* Copy device memory to host */
	hipMemcpy(hostData, devData, n * sizeof(float), hipMemcpyDeviceToHost);
	/* Show result */
	for (i = 0; i < n; i++) {
		if (i % 8 != 0)
			printf(", ");
		else
			printf("\n");

		printf("%1.7f", hostData[i]);
	}
	printf("\n");
	/* Cleanup */
	hiprandDestroyGenerator(gen);
	hipFree(devData);
	free(hostData);

	return EXIT_SUCCESS;
}